#include "hip/hip_runtime.h"
/*

file to test CUDA Kernel

*/
#include <fstream>
#include <stdio.h>
#include <set>
#include <vector>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>

using namespace std;

int *init_zer, *res, *eq, *les;
float *init_inf, *res_f;
const float MAX_LD = 100;

__global__ void addToGridInt(int *G, int h, int b, int xl, int xu, int yl, int yu)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (xl <= i && i < xu && yl <= j && j < yu)
    {
        G[i * b + j] += h;
    }
}

__global__ void addToGridFloat(float *G, float ld, int b, int xl, int xu, int yl, int yu)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (xl <= i && i < xu && yl <= j && j < yu)
    {
        G[i * b + j] += ld;
    }
}

__global__ void updLoadLim(float *G, float v_load, float stress, int b, int xl, int xu, int yl, int yu)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (xl <= i && i < xu && yl <= j && j < yu)
    {
        G[i * b + j] -= stress;
        if (v_load < G[i * b + j])
            G[i * b + j] = v_load;
    }
}

__global__ void checkEq(int *eq, int *G, int val, int b, int xl, int xu, int yl, int yu)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (xl <= i && i < xu && yl <= j && j < yu)
    {
        if (G[i * b + j] == val)
        {
            eq[i * b + j] = 1;
        }
    }
}

__global__ void checkLeq(int *eq, float *G, float val, int b, int xl, int xu, int yl, int yu)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (xl <= i && i < xu && yl <= j && j < yu)
    {
        if (val <= G[i * b + j])
        {
            eq[i * b + j] = 1;
        }
    }
}

class Container
{
private:
    int L;
    int B;

public:
    int *h_grid;
    float *ld_lim;
    // std::set<std::pair<int, int>> pos;
    // std::vector<Item> packedI;

    Container(int L, int B);
    Container(Container &C);
    ~Container();
};

Container::Container(int l, int b)
{
    L = l;
    B = b;

    hipMalloc((void **)&h_grid, L * B * sizeof(int));
    hipMemcpy(h_grid, init_zer, L * B * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void **)&ld_lim, L * B * sizeof(float));
    hipMemcpy(ld_lim, init_inf, L * B * sizeof(float), hipMemcpyHostToDevice);
}

Container::Container(Container &C)
{
    L = C.L;
    B = C.B;
    hipMalloc((void **)&h_grid, L * B * sizeof(int));
    hipMemcpy(h_grid, C.h_grid, L * B * sizeof(int), hipMemcpyDeviceToDevice);

    hipMalloc((void **)&ld_lim, L * B * sizeof(float));
    hipMemcpy(ld_lim, C.ld_lim, L * B * sizeof(float), hipMemcpyDeviceToDevice);
}

Container::~Container()
{
    hipFree(h_grid);
    hipFree(ld_lim);
}

void setup(int l, int b)
{
    fstream f;
    f.open("res1.txt", ios::out);
    f << "";
    f.close();

    init_zer = (int *)malloc(l * b * sizeof(int));
    init_inf = (float *)malloc(l * b * sizeof(float));
    res = (int *)malloc(l * b * sizeof(int));
    res_f = (float *)malloc(l * b * sizeof(float));

    hipMalloc((void **)&eq, l * b * sizeof(int));
    hipMalloc((void **)&les, l * b * sizeof(int));

    for (int i = 0; i < l; i++)
    {
        for (int j = 0; j < b; j++)
        {
            init_zer[i * b + j] = 0;
            init_inf[i * b + j] = MAX_LD;
        }
    }

    return;
}

void deAlloc()
{
    hipFree(les);
    hipFree(eq);

    free(res_f);
    free(res);
    free(init_inf);
    free(init_zer);
}

template <typename Option>
void print2D(Option *res, int l, int b)
{
    fstream f;
    f.open("res1.txt", ios::app);
    for (int i = 0; i < l; i++)
    {
        for (int j = 0; j < b; j++)
        {
            f << res[i * b + j] << "\t";
        }
        f << "\n";
    }
    f << "\n";
    f.close();
}

int main()
{
    int l, b;
    printf("Enter L, B: ");
    scanf("%d %d", &l, &b);

    setup(l, b);

    Container C(l, b);

    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks(1 + l / threadsPerBlock.x, 1 + b / threadsPerBlock.y);

    int h, xl, xu, yl, yu;
    printf("Enter h, xl, xu, yl, yu: ");
    scanf("%d %d %d %d %d", &h, &xl, &xu, &yl, &yu);
    hipDeviceSynchronize();
    addToGridInt<<<numBlocks, threadsPerBlock>>>(C.h_grid, h, b, xl, xu, yl, yu);
    hipDeviceSynchronize();

    hipMemcpy(res, C.h_grid, l * b * sizeof(int), hipMemcpyDeviceToHost);

    print2D<int>(res, l, b);

    float vl, stress;
    printf("Enter vl, stress: ");
    scanf("%f %f", &vl, &stress);
    updLoadLim<<<numBlocks, threadsPerBlock>>>(C.ld_lim, vl, stress, b, xl, xu, yl, yu);
    hipMemcpy(res_f, C.ld_lim, l * b * sizeof(float), hipMemcpyDeviceToHost);

    print2D<float>(res_f, l, b);

    Container C1(C);

    printf("Enter h, xl, xu, yl, yu: ");
    scanf("%d %d %d %d %d", &h, &xl, &xu, &yl, &yu);
    hipDeviceSynchronize();
    addToGridInt<<<numBlocks, threadsPerBlock>>>(C1.h_grid, h, b, xl, xu, yl, yu);

    hipMemcpy(res, C1.h_grid, l * b * sizeof(int), hipMemcpyDeviceToHost);

    print2D<int>(res, l, b);

    printf("Enter vl, stress: ");
    scanf("%f %f", &vl, &stress);
    updLoadLim<<<numBlocks, threadsPerBlock>>>(C1.ld_lim, vl, stress, b, xl, xu, yl, yu);
    hipMemcpy(res_f, C1.ld_lim, l * b * sizeof(float), hipMemcpyDeviceToHost);

    print2D<float>(res_f, l, b);

    int val;
    printf("Enter val xl, xu, yl, yu: ");
    scanf("%d %d %d %d %d", &val, &xl, &xu, &yl, &yu);
    hipMemcpy(eq, init_zer, l * b * sizeof(int), hipMemcpyHostToDevice);
    checkEq<<<numBlocks, threadsPerBlock>>>(eq, C1.h_grid, val, b, xl, xu, yl, yu);
    hipMemcpy(res, eq, l * b * sizeof(int), hipMemcpyDeviceToHost);
    print2D<int>(res, l, b);

    thrust::device_ptr<int> dev_ptr(eq);
    int total = thrust::reduce(thrust::device, dev_ptr, dev_ptr + (l * b), 0);
    printf("%d %d", total, (xu - xl) * (yu - yl));

    float load;
    printf("Enter load, xl, xu, yl, yu: ");
    scanf("%f %d %d %d %d", &load, &xl, &xu, &yl, &yu);
    hipMemcpy(eq, init_zer, l * b * sizeof(int), hipMemcpyHostToDevice);
    checkLeq<<<numBlocks, threadsPerBlock>>>(eq, C1.ld_lim, load, b, xl, xu, yl, yu);
    hipMemcpy(res, eq, l * b * sizeof(int), hipMemcpyDeviceToHost);
    print2D<int>(res, l, b);

    total = thrust::reduce(thrust::device, dev_ptr, dev_ptr + (l * b), 0);
    printf("%d %d", total, (xu - xl) * (yu - yl));

    deAlloc();

    return 0;
}
