#include "hip/hip_runtime.h"
/*

Packing File

*/

#include <stdio.h>

#include <iostream>

#include <fstream>

#include <algorithm>

#include <set>

#include <vector>

#include <thrust/reduce.h>

#include <thrust/device_ptr.h>

#include <thrust/execution_policy.h>

using namespace std;

const int TREE_WIDTH = 5;
int* init_zer, * res, * eq, * les;
float* init_inf, * res_f;
const float MAX_LD = 1;

bool DEBUG = true, DEBUG_3D = false;

char tmp;

template < typename T >
void print2D(T* res, int l, int b) {
    fstream f("res.txt", ios::out);
    f << '\t';
    for (int j = 0; j < b; j++) {
        f << j << '\t';
    }
    f << '\n';
    for (int i = 0; i < l; i++) {
        f << i << '\t';
        for (int j = 0; j < b; j++) {
            f << res[i * b + j] << "\t";
        }
        f << "\n";
    }
    f << "\n";
}

__global__ void addToGridInt(int* G, int val, int b, int xl, int xu, int yl, int yu) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (xl <= i && i < xu && yl <= j && j < yu) {
        G[i * b + j] += val;
    }
}

__global__ void updLoadLim(float* G, float ld_lim, float load, int b, int xl, int xu, int yl, int yu) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (xl <= i && i < xu && yl <= j && j < yu) {
        G[i * b + j] -= load;
        if (ld_lim < G[i * b + j])
            G[i * b + j] = ld_lim;
    }
}

__global__ void checkEq(int* eq, int* G, int val, int b, int xl, int xu, int yl, int yu) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (xl <= i && i < xu && yl <= j && j < yu) {
        if (G[i * b + j] == val) {
            eq[i * b + j] = 1;
        }
    }
}

__global__ void checkLeq(int* eq, float* G, float val, int b, int xl, int xu, int yl, int yu) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (xl <= i && i < xu && yl <= j && j < yu) {
        if (val <= G[i * b + j]) {
            eq[i * b + j] = 1;
        }
    }
}

struct Location {
    int x, y, z;
    Location() {
        x = -1, y = -1, z = -1;
    }

    Location(int X, int Y, int Z) {
        x = X, y = Y, z = Z;
    }

    Location(Location& L) {
        x = L.x, y = L.y, z = L.z;
    }
};

struct Item {
    int id, dst;
    float wt;
    int l, b, h;
    long vol;
    int l1, b1, h1;

    int valid;

    vector < int > orientation;
    vector < float > stackload;

    int max_dim;
    float v_ld_lim;
    bool packed;
    Location pos;

    Item();
    Item(int, int, float, int, int, int, vector < int >, vector < float >);
    Item(Item& I);

    bool operator < (const Item& I) const {
        if (dst != I.dst) {
            return dst < I.dst;
        }
        float m1 = 0, m2 = 0;
        for (int i = 0; i < 3; i++) {
            m1 = max(m1, (float)orientation[i] * stackload[i]);
            m2 = max(m2, (float)I.orientation[i] * I.stackload[i]);
        }
        if (m1 != m2) {
            return m1 < m2;
        }
        return id < I.id;
    }

    float stress_load() {
        float base_area = l1 * b1;
        return wt / base_area;
    }

    void setPackDim(int lo, int bo, int ho) {
        l1 = lo;
        b1 = bo;
        h1 = ho;
    }

    void printObj() {
        printf("%d %d %.3f\n", id, dst, wt);
        printf("%d %d %d\n", l, b, h);
        printf("%d %d %d\n", orientation[0], orientation[1], orientation[2]);
        printf("%.3f %.3f %.3f, v: %3f\n", stackload[0], stackload[1], stackload[2], v_ld_lim);
        //printf("\n");
    }
};

Item::Item() {
    id = 0, dst = 0, wt = 0;
    l = 0, b = 0, h = 0;
    l1 = 0, b1 = 0, h1 = 0;
    vol = 0;
    max_dim = 0;
    v_ld_lim = 0;
    packed = 0;
    valid = 1;
    pos = Location();
}

Item::Item(int Id, int Dst, float Wt, int L, int B, int H, vector < int > ornt, vector < float > stld) {
    id = Id, dst = Dst, wt = Wt;
    l = L, b = B, h = H;
    l1 = l, b1 = b, h1 = h;

    vol = l * 1.0 * b * 1.0 * h;

    stackload = stld;
    orientation = ornt;

    max_dim = max({
        l,
        b,
        h
        });
    v_ld_lim = stackload[2];

    packed = 0;
    valid = 1;
    pos = Location();
}

Item::Item(Item& I) {
    id = I.id, dst = I.dst, wt = I.wt;
    l = I.l, b = I.b, h = I.h;
    l1 = I.l1, b1 = I.b1, h1 = I.h1;

    vol = l * 1.0 * b * 1.0 * h;

    stackload = I.stackload;
    orientation = I.orientation;

    max_dim = I.max_dim;
    v_ld_lim = I.v_ld_lim;
    packed = I.packed;
    valid = I.valid;
    pos = Location(I.pos);
}

struct Container {
    int L;
    int B;
    int H;

    float vol, util_vol;

    int* h_grid;
    float* ld_lim;

    std::set < std::pair < int, int >> corners;
    std::vector < Item > packedI;

    Container();
    Container(int, int, int);
    Container(Container& C);
    ~Container();

    Location fit(int, int, int, float);
    float volUtil() {
        return (float)util_vol / (float)vol;
    }

    int itemCount() {
        return packedI.size();
    }

    void printObj() {
        printf("%d %d %d\n", L, B, H);
        printf("%.3f %.3f\n", vol, util_vol);
        printf("\n");
    }

};

Container::Container() {
    L = 0, B = 0, H = 0;
    vol = 0, util_vol = 0;
}

Container::Container(int l, int b, int h) {
    L = l;
    B = b;
    H = h;

    vol = L * 1.0 * B * 1.0 * H;
    util_vol = 0;

    corners.insert({
        0,
        0
        });

    hipMalloc((void**)&h_grid, L * B * sizeof(int));
    hipMemcpy(h_grid, init_zer, L * B * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void**)&ld_lim, L * B * sizeof(float));
    hipMemcpy(ld_lim, init_inf, L * B * sizeof(float), hipMemcpyHostToDevice);
}

Container::Container(Container& C) {
    L = C.L;
    B = C.B;
    H = C.H;

    vol = L * 1.0 * B * 1.0 * H;
    util_vol = C.util_vol;

    corners = C.corners;

    packedI = vector < Item >();
    for (Item& i : C.packedI) {
        packedI.push_back(Item(i));
    }

    hipMalloc((void**)&h_grid, L * B * sizeof(int));
    hipMemcpy(h_grid, C.h_grid, L * B * sizeof(int), hipMemcpyDeviceToDevice);

    hipMalloc((void**)&ld_lim, L * B * sizeof(float));
    hipMemcpy(ld_lim, C.ld_lim, L * B * sizeof(float), hipMemcpyDeviceToDevice);
}

Container::~Container() {
    hipFree(h_grid);
    hipFree(ld_lim);
}

Location Container::fit(int l, int b, int h, float load) {
    if (DEBUG_3D) {
        cout << "Inside Fit: ";
        // cout << l << ' ' << b << ' ' << h << ' ' << load << '\n';
    }
    Location loc;

    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks(1 + L / threadsPerBlock.x, 1 + B / threadsPerBlock.y);

    for (auto p : corners) {
        if (DEBUG_3D) {
            cout << p.first << " " << p.second << '\n';
        }

        int pos_valid = 1;
        int x = p.first, y = p.second;
        hipMemcpy(res, h_grid, L * B * sizeof(int), hipMemcpyDeviceToHost);
        int base = res[x * B + y];

        if (x + l > L || y + b > B || base + h > H) {
            pos_valid = 0;
            continue;
        }

        int xl = x, xu = x + l;
        int yl = y, yu = y + b;
        int area = (xu - xl) * (yu - yl);
        int total;

        if (DEBUG_3D) {
            cout << xl << ' ' << xu << ' ';
            cout << yl << ' ' << yu << ' ';
            cout << area << '\n';
            cout << "Base:" << base << '\n';
        }

        hipMemcpy(eq, init_zer, L * B * sizeof(int), hipMemcpyHostToDevice);
        hipDeviceSynchronize();
        checkEq << < numBlocks, threadsPerBlock >> > (eq, h_grid, base, B, xl, xu, yl, yu);
        thrust::device_ptr < int > dev_ptr(eq);
        total = thrust::reduce(thrust::device, dev_ptr, dev_ptr + (L * B), 0);
        if (DEBUG_3D) {
            cout << "H tot:" << total << '\n';

            hipMemcpy(res, eq, L * B * sizeof(int), hipMemcpyDeviceToHost);
            print2D<int>(res, L, B);
            cout << "Eq of H in file: ";
            cin >> tmp;

            hipMemcpy(res, h_grid, L * B * sizeof(int), hipMemcpyDeviceToHost);
            print2D<int>(res, L, B);
            cout << "H in file: ";
            cin >> tmp;
        }
        if (total != area) {
            pos_valid = 0;
            continue;
        }

        hipMemcpy(les, init_zer, L * B * sizeof(int), hipMemcpyHostToDevice);
        hipDeviceSynchronize();
        checkLeq << < numBlocks, threadsPerBlock >> > (les, ld_lim, load, B, xl, xu, yl, yu);
        thrust::device_ptr < int > dev_ptr2(les);
        total = thrust::reduce(thrust::device, dev_ptr2, dev_ptr2 + (L * B), 0);
        if (DEBUG_3D) {
            cout << "L tot:" << total << '\n';

            hipMemcpy(res, les, L * B * sizeof(int), hipMemcpyDeviceToHost);
            print2D<int>(res, L, B);
            cout << "Les of L in file: ";
            cin >> tmp;

            hipMemcpy(res_f, ld_lim, L * B * sizeof(int), hipMemcpyDeviceToHost);
            print2D<float>(res_f, L, B);
            cout << "L in file: ";
            cin >> tmp;
        }
        if (total != area) {
            pos_valid = 0;
            continue;
        }

        if (pos_valid) {
            loc = Location(x, y, base);
            break;
        }
    }

    if (DEBUG_3D) {
        cout << loc.x << ' ' << loc.y << ' ' << loc.z << ": ";
        cin >> tmp;
        cout << '\n';
    }
    return loc;
}

struct State {
    Container C;
    float g;

    bool operator < (const State& t) const {
        if (g != t.g) {
            return g < t.g;
        }
        return C.packedI.size() > t.C.packedI.size();
    }

    State() {
        g = 0;
        C = Container();
    }
    State(float g, Container C) {
        this->g = g;
        this->C = C;
    }

    State(State& O) {
        C = O.C;
        g = O.g;
    }

    void printObj() {
        cout << "g: " << g << '\n';
        cout << "C: ";
        C.printObj();
    }
};

void setup(int L, int B) {

    init_zer = (int*)malloc(L * B * sizeof(int));
    init_inf = (float*)malloc(L * B * sizeof(float));
    res = (int*)malloc(L * B * sizeof(int));
    res_f = (float*)malloc(L * B * sizeof(float));

    hipMalloc((void**)&eq, L * B * sizeof(int));
    hipMalloc((void**)&les, L * B * sizeof(int));

    for (int i = 0; i < L; i++) {
        for (int j = 0; j < B; j++) {
            init_zer[i * B + j] = 0;
            init_inf[i * B + j] = MAX_LD;
        }
    }

    if (DEBUG) {
        cout << "set up done\n\n";
    }

    return;
}

void deAlloc() {
    hipFree(les);
    hipFree(eq);

    free(res_f);
    free(res);
    free(init_inf);
    free(init_zer);
}

vector < Item > allowedOrientations(Item& I) {
    // if (DEBUG) {
    //     cout << "\tInside Allowed Orientations\n";
    //     I.printObj();
    // }
    vector < Item > res(6);
    if (I.orientation[1] == 1) {
        res[0] = Item(I);
        res[0].setPackDim(I.h, I.b, I.l);
        res[0].v_ld_lim = I.stackload[0];

        res[1] = Item(I);
        res[1].setPackDim(I.b, I.h, I.l);
        res[1].v_ld_lim = I.stackload[0];
    }
    else {
        res[0].valid = 0;
        res[1].valid = 0;
    }

    if (I.orientation[1] == 1) {
        res[2] = Item(I);
        res[2].setPackDim(I.l, I.h, I.b);
        res[2].v_ld_lim = I.stackload[1];

        res[3] = Item(I);
        res[3].setPackDim(I.h, I.l, I.b);
        res[3].v_ld_lim = I.stackload[1];
    }
    else {
        res[2].valid = 0;
        res[3].valid = 0;
    }

    if (I.orientation[2] == 1) {
        res[4] = Item(I);
        res[4].setPackDim(I.l, I.b, I.h);
        res[4].v_ld_lim = I.stackload[2];

        res[5] = Item(I);
        res[5].setPackDim(I.b, I.l, I.h);
        res[5].v_ld_lim = I.stackload[2];
    }
    else {
        res[4].valid = 0;
        res[5].valid = 0;
    }

    // if (DEBUG) {
    //     cout << "\tExiting Allowed Orientations\n";
    // }

    return res;
}

void packItem(Container& C, Item& I) {
    if (DEBUG) {
        // cout << "Inside Pack Item\n";
    }
    if (I.pos.x == -1) {
        return;
    }
    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks(1 + C.L / threadsPerBlock.x, 1 + C.B / threadsPerBlock.y);

    // updating h_grid;
    int xl = I.pos.x, xu = I.pos.x + I.l1;
    int yl = I.pos.y, yu = I.pos.y + I.b1;
    if (DEBUG) {
        // cout << xl << ' ' << xu << ' ' << yl << ' ' << yu << '\n';
    }
    hipDeviceSynchronize();
    addToGridInt << < numBlocks, threadsPerBlock >> > (C.h_grid, I.h1, C.B, xl, xu, yl, yu);
    // if (DEBUG) {
    //     cout << I.h1 << '\n';
    //     hipMemcpy(res, C.h_grid, C.L * C.B * sizeof(int), hipMemcpyDeviceToHost);
    //     print2D<int>(res, C.L, C.B);
    //     cout << "H in file: ";
    //     cin >> tmp;
    // }
    // updating ld_lim;
    float load = I.stress_load();
    float ld_lim = I.v_ld_lim;
    hipDeviceSynchronize();
    updLoadLim << < numBlocks, threadsPerBlock >> > (C.ld_lim, ld_lim, load, C.B, xl, xu, yl, yu);
    // if (DEBUG) {
    //     cout << ld_lim << ' ' << load << '\n';
    //     hipMemcpy(res_f, C.ld_lim, C.L * C.B * sizeof(int), hipMemcpyDeviceToHost);
    //     print2D<float>(res_f, C.L, C.B);
    //     cout << "L in file: ";
    //     cin >> tmp;
    // }

    if (I.pos.x + I.l1 < C.L) {
        C.corners.insert({
            I.pos.x + I.l1,
            I.pos.y
            });
    }
    if (I.pos.y + I.b1 < C.B) {
        C.corners.insert({
            I.pos.y,
            I.pos.y + I.b1
            });
    }
    if (I.pos.z + I.h1 < C.H) {
        C.corners.insert({
            I.pos.x,
            I.pos.y
            });
    }

    I.packed = 1;
    C.packedI.push_back(Item(I));
    C.util_vol += I.vol;

    if (DEBUG) {
        // cout << "Exiting Pack Item\n\n";
    }

    return;
}

float greedyPack(Container C, vector < Item >& items, int start) {
    if (DEBUG) {
        // cout << "Inside Greedy\n";
    }
    for (int i = start; i >= 0; i--) {
        if (DEBUG) {
            // cout << i << '\n';
        }
        Item I = items[i];
        vector < Item > Iarr = allowedOrientations(I);

        for (int j = 0; j < 6; j++) {
            if (DEBUG) {
                // cout << j << '\n';
            }
            if (Iarr[j].valid == 0) {
                continue;
            }
            Iarr[j].pos = C.fit(Iarr[j].l1, Iarr[j].b1, Iarr[j].h1, Iarr[j].stress_load());

            if (Iarr[j].pos.x != -1) {
                packItem(C, Iarr[j]);
                break;
            }
        }
    }

    if (DEBUG) {
        cout << C.volUtil() << "\n\n";
    }

    if (DEBUG) {
        // cout << "Exiting Greedy\n\n";
    }
    return C.volUtil();
}

bool greater_pair(std::pair < float, Container >& a, std::pair < float, Container >& b) {
    if (a.first > b.first) {
        return true;
    }
    else if (b.first > a.first) {
        return false;
    }
    else {
        return a.second.itemCount() <= b.second.itemCount(); // more larger packages more likely to have been packed in packing with less overall packages
    }
}

int main(int argc, char* argv[]) {
    // Reading Inputs
    int L, B, H, n;
    fstream f;
    f.open(argv[1], ios::in);
    f >> L >> B >> H >> n;

    printf("%d %d %d %d\n", L, B, H, n);

    int id, dst;
    float wt;
    int l, b, h;
    vector < int > ornt(3);
    vector < float > stld(3);

    vector < Item > items;
    for (int i = 0; i < n; i++) {
        f >> id >> dst >> wt;
        f >> l >> b >> h;
        f >> ornt[0] >> ornt[1] >> ornt[2];
        f >> stld[0] >> stld[1] >> stld[2];
        items.push_back(Item(id, dst, wt, l, b, h, ornt, stld));
        if (DEBUG) {
            // cout << items[i].id << ' ' << items[i].v_ld_lim << '\n';
        }
    }
    f.close();

    // Setup
    setup(L, B);

    //Container Object
    Container C(L, B, H);
    C.printObj();

    std::sort(items.begin(), items.end());

    // if (DEBUG) {
    //     for (Item I : items) {
    //         I.printObj();
    //     }
    // }

    float iVol = 0;
    for (Item& i : items) {
        iVol += i.vol;
    }

    printf("%f\n\n", iVol / C.vol);

    vector < State > tree;

    float g = greedyPack(C, items, n - 1);
    State S(g, C);
    if (DEBUG) {
        //S.printObj();
    }
    tree.push_back(State(S));
    if (DEBUG) {
        cout << "Tree len: " << tree.size() << '\n';
        // tree[0].printObj();
    }

    for (int i = n - 1; i >= 0; i--) {
        if (DEBUG) {
            cout << i << '\n';
        }
        Item I = items[i];
        if (DEBUG) {
            //I.printObj();
        }

        vector < Item > Iarr = allowedOrientations(I);

        for (int k = tree.size() - 1; k >= 0; k--) {
            Container C_state(tree[k].C);

            if (DEBUG) {
                // cout << k << '\n';
                // tree[k].printObj();
            }
            cout << "Empty\n"; 
            Container C1(C_state);
            cout << C1.volUtil() << '\n';
            tree.push_back(State(greedyPack(C1, items, i - 1), C1));

            for (int j = 0; j < 6; j++) {
                if (Iarr[j].valid == 0) {
                    continue;
                }
                if (DEBUG) {
                    cout << j << '\n';
                }

                Container C_new(C_state);
                cout << C_new.volUtil() << '\n';

                if (DEBUG) {
                    cout << "Tree Size: " << tree.size() << " k: " << k << '\n';
                    hipMemcpy(res, tree[k].C.h_grid, L * B * sizeof(int), hipMemcpyDeviceToHost);
                    print2D<int>(res, L, B);
                    cout << "Pre fit tree[k].C H in file: ";
                    cin >> tmp;

                    hipMemcpy(res_f, tree[k].C.ld_lim, L * B * sizeof(int), hipMemcpyDeviceToHost);
                    print2D<float>(res_f, L, B);
                    cout << "Pre fit tree[k].C L in file: ";
                    cin >> tmp;

                    hipMemcpy(res, C_new.h_grid, L * B * sizeof(int), hipMemcpyDeviceToHost);
                    print2D<int>(res, L, B);
                    cout << "Pre fit C_new H in file: ";
                    cin >> tmp;

                    hipMemcpy(res_f, C_new.ld_lim, L * B * sizeof(int), hipMemcpyDeviceToHost);
                    print2D<float>(res_f, L, B);
                    cout << "Pre fit C_new L in file: ";
                    cin >> tmp;
                }

                DEBUG_3D = true;
                Iarr[j].pos = C_new.fit(Iarr[j].l1, Iarr[j].b1, Iarr[j].h1, Iarr[j].stress_load());
                DEBUG_3D = false;

                if (Iarr[j].pos.x != -1) {
                    if (DEBUG) {
                        for (auto t : tree) {
                            cout << t.C.volUtil() << ' ';
                        }
                        cout << '\n';
                        
                        cout << "Tree Size: " << tree.size() << " k: " << k << '\n';
                        hipMemcpy(res, tree[k].C.h_grid, L * B * sizeof(int), hipMemcpyDeviceToHost);
                        print2D<int>(res, L, B);
                        cout << "Pre Pack tree[k].C H in file: ";
                        cin >> tmp;

                        hipMemcpy(res_f, tree[k].C.ld_lim, L * B * sizeof(int), hipMemcpyDeviceToHost);
                        print2D<float>(res_f, L, B);
                        cout << "Pre Pack tree[k].C L in file: ";
                        cin >> tmp;
                    }
                    packItem(C_new, Iarr[j]);

                    if (DEBUG) {
                        for (auto t : tree) {
                            cout << t.C.volUtil() << ' ';
                        }
                        cout << '\n';

                        cout << "Tree Size: " << tree.size() << " k: " << k << '\n';
                        hipMemcpy(res, tree[k].C.h_grid, L * B * sizeof(int), hipMemcpyDeviceToHost);
                        print2D<int>(res, L, B);
                        cout << "Post Pack tree[k].C H in file: ";
                        cin >> tmp;

                        hipMemcpy(res_f, tree[k].C.ld_lim, L * B * sizeof(int), hipMemcpyDeviceToHost);
                        print2D<float>(res_f, L, B);
                        cout << "Post Pack tree[k].C L in file: ";
                        cin >> tmp;
                    }

                    tree.push_back(State(greedyPack(C_new, items, i - 1), C_new));
                }
            }
            tree.erase(tree.begin() + k);
        }

        std::sort(tree.begin(), tree.end());

        cout << n - i << ": ";
        for (State s : tree) {
            string st = to_string(s.g);
            if (st.size() < 5) {
                st += "0";
            }
            cout << st << " ";
        }
        cout << "\n";

        if (tree.size() > TREE_WIDTH) {
            tree.resize(TREE_WIDTH);
        }

        cout << "After Resizing:\n";
        cout << n - i << ": ";
        for (State s : tree) {
            string st = to_string(s.g);
            if (st.size() < 5) {
                st += "0";
            }
            cout << st << " ";
        }
        cout << "\n\n";
    }

    Container resC = tree[0].C;
    printf("%f", resC.volUtil());

    deAlloc();

    return 0;
}